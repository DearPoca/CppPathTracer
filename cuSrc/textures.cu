#include "hip/hip_runtime.h"
#include "textures.h"

#include <hip/hip_runtime.h>
#include <helper_functions.h>  
#include <hip/hip_runtime_api.h>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgproc/types_c.h>
#include <opencv2/imgcodecs/legacy/constants_c.h>

#include "logger.hpp"

hipTextureObject_t AddTexByFile(std::string file_path, hipTextureAddressMode addr_mode, hipTextureFilterMode filter_mode) {
	cv::Mat src = cv::imread(file_path, CV_LOAD_IMAGE_GRAYSCALE);

	int width = src.cols;
	int height = src.rows;

	hipError_t err;

	float* h_data = (float*)std::malloc(sizeof(float) * width * height);
	for (int i = 0; i < height * width; ++i)
		h_data[i] = i;

	hipChannelFormatDesc channelDesc =
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray_t cuArray;
	err = hipMallocArray(&cuArray, &channelDesc, width, height);
	if (err != hipSuccess)
	{
		log_error("Error occur with AddTexByFile: %s", hipGetErrorString(err));
	}

	// Set pitch of the source (the width in memory in bytes of the 2D array pointed
	// to by src, including padding), we dont have any padding
	const size_t spitch = width * sizeof(float);
	// Copy data located at address h_data in host memory to device memory
	err = hipMemcpy2DToArray(cuArray, 0, 0, h_data, spitch, width * sizeof(float),
		height, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		log_error("Error occur with AddTexByFile: %s", hipGetErrorString(err));
	}

	// Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	// Specify texture object parameters
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = addr_mode;
	texDesc.addressMode[1] = addr_mode;
	texDesc.filterMode = filter_mode;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	// Create texture object
	hipTextureObject_t texObj = 0;
	err = hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
	if (err != hipSuccess)
	{
		log_error("Error occur with AddTexByFile: %s", hipGetErrorString(err));
	}

	return texObj;
}

__device__ float4 GetTexture2D(hipTextureObject_t tex_obj, float u, float v) {
	float4 tex = tex2D<float4>(tex_obj, u, v);
	return tex;
}