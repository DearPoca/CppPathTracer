#include "hip/hip_runtime.h"
#include "textures.h"

#include <hip/hip_runtime.h>
#include <helper_functions.h>  
#include <hip/hip_runtime_api.h>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgproc/types_c.h>


hipTextureObject_t AddTexByFile(std::string file_path, hipTextureAddressMode addr_mode = hipAddressModeMirror, hipTextureFilterMode filter_mode = hipFilterModeLinear) {
	cv::Mat src = cv::imread(file_path);
	cv::Mat dst;
	cv::cvtColor(src, dst, CV_BGR2RGBA);

	int width = dst.cols;
	int height = dst.rows;

	hipArray* cu_array;
	hipChannelFormatDesc cu_desc = hipCreateChannelDesc<uchar4>();
	checkCudaErrors(hipMallocArray(&cu_array, &cu_desc, width, height));
	checkCudaErrors(hipMemcpy2DToArray(cu_array, 0, 0, dst.data, width * 4, width * sizeof(uint8_t), height,
		hipMemcpyHostToDevice));

	hipResourceDesc res_desc;
	memset(&res_desc, 0, sizeof(res_desc));
	res_desc.resType = hipResourceTypeArray;
	res_desc.res.array.array = cu_array;

	hipTextureDesc tex_desc;
	tex_desc.addressMode[0] = addr_mode;
	tex_desc.addressMode[1] = addr_mode;
	tex_desc.filterMode = filter_mode;
	tex_desc.readMode = hipReadModeNormalizedFloat;
	tex_desc.normalizedCoords = 1;

	hipTextureObject_t tex_object;

	checkCudaErrors(hipCreateTextureObject(&tex_object, &res_desc, &tex_desc, NULL));
	return tex_object;
}

__device__ float4 GetTexture2D(hipTextureObject_t tex_obj, float u, float v) {
	float4 tex = tex2D<float4>(tex_obj, u, v);
	return tex;
}