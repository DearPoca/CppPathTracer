#include "hip/hip_runtime.h"
#include "textures.h"

#include <hip/hip_runtime.h>
#include <helper_functions.h>  
#include <hip/hip_runtime_api.h>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgproc/types_c.h>
#include <opencv2/imgcodecs/legacy/constants_c.h>

#include "logger.hpp"

hipTextureObject_t PocaTextureUtils::AddTexByFile(std::string file_path, hipTextureAddressMode addr_mode, hipTextureFilterMode filter_mode) {
	cv::Mat src = cv::imread(file_path);
	cv::Mat dst;
	cv::cvtColor(src, dst, CV_BGR2RGBA);

	int width = dst.cols;
	int height = dst.rows;

	hipError_t err;

	hipArray* cu_array;
	hipChannelFormatDesc cu_desc = hipCreateChannelDesc<uchar4>();
	err = hipMallocArray(&cu_array, &cu_desc, width, height);
	if (err != hipSuccess)
	{
		log_error("Error occur with AddTexByFile: %s", hipGetErrorString(err));
	}

	err = hipMemcpy2DToArray(cu_array, 0, 0, dst.data, width * 4, width * sizeof(uint8_t), height,
		hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		log_error("Error occur with AddTexByFile: %s", hipGetErrorString(err));
	}

	hipResourceDesc res_desc;
	memset(&res_desc, 0, sizeof(res_desc));
	res_desc.resType = hipResourceTypeArray;
	res_desc.res.array.array = cu_array;

	hipTextureDesc tex_desc;
	memset(&tex_desc, 0, sizeof(hipTextureDesc));
	tex_desc.addressMode[0] = addr_mode;
	tex_desc.addressMode[1] = addr_mode;
	tex_desc.filterMode = filter_mode;
	tex_desc.readMode = hipReadModeNormalizedFloat;
	tex_desc.normalizedCoords = 1;

	hipTextureObject_t tex_object;

	err = hipCreateTextureObject(&tex_object, &res_desc, &tex_desc, nullptr);
	if (err != hipSuccess)
	{
		log_error("Error occur with AddTexByFile: %s", hipGetErrorString(err));
	}

	log_info("Add texture, path: %s, obj_t: %ld", file_path.c_str(), tex_object);
	return tex_object;
}

void PocaTextureUtils::DestroyTexture(hipTextureObject_t tex) {
	hipDestroyTextureObject(tex);
}

__device__ float4 PocaTextureUtils::GetTexture2D(hipTextureObject_t tex_obj, float u, float v) {
	float4 tex = tex2D<float4>(tex_obj, u, v);
	return tex;
}