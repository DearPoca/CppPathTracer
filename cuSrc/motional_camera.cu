#include "hip/hip_runtime.h"

#include "motional_camera.h"

#include <hip/hip_runtime.h>
#include <helper_functions.h>  
#include <hip/hip_runtime_api.h>
#include <mutex>

#include "ray_tracing_math.hpp"
#include "ray_tracing_common.h"

std::mutex camera_mutex;

MotionalCamera::MotionalCamera() :
	width_(1920),
	height_(1080),
	origin_(make_float3(0.f)),
	look_at_(make_float3(0.f, 0.f, 1.f)),
	cur_sample_idx_(0) {

}

MotionalCamera::MotionalCamera(int width, int height) :
	width_(width),
	height_(height),
	origin_(make_float3(0.f)),
	look_at_(make_float3(0.f, 0.f, 1.f)),
	cur_sample_idx_(0) {

}

MotionalCamera::MotionalCamera(int width, int height, float3 ori, float3 at) :
	width_(width),
	height_(height),
	origin_(ori),
	look_at_(at),
	cur_sample_idx_(0) {

}

MotionalCamera::~MotionalCamera() {}

void MotionalCamera::Refresh() {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);
	cur_sample_idx_ = 0;
}

void MotionalCamera::SetViewFov(float fov) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);
	view_fov_ = fov;
}

void MotionalCamera::Resize(int width, int height) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);
	width_ = width;
	height_ = height;
}

void MotionalCamera::SetOrigin(float3 ori) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);
	origin_ = ori;
}

void MotionalCamera::SetOrigin(float x, float y, float z) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);
	origin_.x = x;
	origin_.y = y;
	origin_.z = z;
}

void MotionalCamera::SetLookAt(float3 look_at) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);
	look_at_ = look_at;
}

void MotionalCamera::SetLookAt(float x, float y, float z) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);
	look_at_.x = x;
	look_at_.y = y;
	look_at_.z = z;
}

void MotionalCamera::MoveEyeLeft(float coefficient) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	float3 w = normalize(origin_ - look_at_);
	float3 left = normalize(cross(vup, w));

	origin_ += coefficient * move_speed_ * left;
	look_at_ += coefficient * move_speed_ * left;
}

void MotionalCamera::MoveEyeRight(float coefficient) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	float3 w = normalize(origin_ - look_at_);
	float3 left = normalize(cross(vup, w));

	origin_ -= coefficient * move_speed_ * left;
	look_at_ -= coefficient * move_speed_ * left;
}

void MotionalCamera::MoveEyeForward(float coefficient) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	float3 w = normalize(origin_ - look_at_);
	float3 left = normalize(cross(vup, w));
	float3 back = normalize(cross(left, vup));

	origin_ -= coefficient * move_speed_ * back;
	look_at_ -= coefficient * move_speed_ * back;
}

void MotionalCamera::MoveEyeBackward(float coefficient) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	float3 w = normalize(origin_ - look_at_);
	float3 left = normalize(cross(vup, w));
	float3 back = normalize(cross(left, vup));

	origin_ += coefficient * move_speed_ * back;
	look_at_ += coefficient * move_speed_ * back;
}

void MotionalCamera::MoveEyeUp(float coefficient) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	origin_ += coefficient * move_speed_ * vup;
	look_at_ += coefficient * move_speed_ * vup;
}

void MotionalCamera::MoveEyeDown(float coefficient) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	origin_ -= coefficient * move_speed_ * vup;
	look_at_ -= coefficient * move_speed_ * vup;
}

void MotionalCamera::RotateAroundUp(float dy) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	look_at_ = origin_ + normalize(look_at_ - origin_);
	look_at_ += dy * vup;
	look_at_ = origin_ + normalize(look_at_ - origin_);
}

void MotionalCamera::RotateAroundDown(float dy) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	look_at_ = origin_ + normalize(look_at_ - origin_);
	look_at_ -= dy * vup;
	look_at_ = origin_ + normalize(look_at_ - origin_);
}

void MotionalCamera::RotateAroundLeft(float dx) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	look_at_ = origin_ + normalize(look_at_ - origin_);

	float3 w = normalize(origin_ - look_at_);
	float3 left = normalize(cross(vup, w));

	look_at_ += dx * left;
	look_at_ = origin_ + normalize(look_at_ - origin_);
}

void MotionalCamera::RotateAroundRight(float dx) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	look_at_ = origin_ + normalize(look_at_ - origin_);

	float3 w = normalize(origin_ - look_at_);
	float3 left = normalize(cross(vup, w));
	float3 back = normalize(cross(left, vup));

	look_at_ -= dx * left;
	look_at_ = origin_ + normalize(look_at_ - origin_);
}

void MotionalCamera::ScaleFov(float d) {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	view_fov_ += d * M_PI / 180.0f;
}

MotionalCamera MotionalCamera::GetCopy() {
	std::lock_guard<std::mutex> lock_guard(camera_mutex);

	float theta = view_fov_ * M_PI / 180;
	float aspectRatio = float(width_) / float(height_);
	float half_height = tan(theta / 2);
	float half_width = aspectRatio * half_height;

	w_ = normalize(origin_ - look_at_);
	u_ = normalize(cross(vup, w_));
	v_ = cross(w_, u_);

	dist_to_focus_ = length(origin_ - look_at_);

	top_left_corner_ =
		origin_ - half_width * dist_to_focus_ * u_ + half_height * dist_to_focus_ * v_ - dist_to_focus_ * w_;
	horizontal_ = 2 * half_width * dist_to_focus_ * u_;
	vertical_ = -2 * half_height * dist_to_focus_ * v_;

	cur_sample_idx_++;

	MotionalCamera ret;
	ret = *this;
	return ret;
}

__device__ Ray MotionalCamera::RayGen(int x, int y, hiprandState& state) {
	Ray ray;
	float3 rd = lens_radius_ * device_create_random_float3(state);
	float3 offset = u_ * rd.x + v_ * rd.y;
	float dx = float(x) / float(width_);
	float dy = float(y) / float(height_);
	ray.origin = origin_ + offset;
	ray.dir = normalize(top_left_corner_ + dx * horizontal_ + dy * vertical_ - origin_ - offset);
	ray.tmin = 0.f;
	ray.tmax = DEFAULT_RAY_TMAX;
	return ray;
}
